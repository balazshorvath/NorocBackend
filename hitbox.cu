
#include <hip/hip_runtime.h>
extern "C"

__global__ void add(int n, float *sum) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        sum[i] = i;
    }
}
